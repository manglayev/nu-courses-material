#include "hip/hip_runtime.h"
#include "kernel_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void cudaFunction(int *d)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if(index < 1)
  {
    printf("1. d[%d] = %d\n", 0, d[0]);
    d[0] = d[0] * 3;
    printf("2. d[%d] = %d\n", 0, d[0]);
  }
}

void wrapper(int c)
{
  printf("BEFORE:\n");
  printf("c = %d;\n", c);
  int *dev_c;
  hipMallocManaged(&dev_c, sizeof(int));
  dev_c[0] = c;
  cudaFunction<<<BLOCKS, THREADS>>>(dev_c);
  hipDeviceSynchronize();
  printf("AFTER:\n");
  printf("dev_c[%d] = %d;\n", 0, dev_c[0]);
}
