#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>

#define THREADS 1
#define BLOCKS 1

template<typename T> __host__ __device__ T signum(T x)
{
  if(x > 0)
    return 1;
  else if(x < 0)
    return -1;
  else
    return 0;
}

__global__ void signumCaller(int *x)
{
  printf("1 received in kernel = %d\n", x[0]);
  x[0] = x[0] * (-1);
  printf("2 changed in kernel = %d\n", x[0]);
  x[0] = signum(x[0]);
  printf("3 sending from kernel = %d\n", x[0]);
}

int main()
{
  int a = 5;
  float b = -19.234;
  int c = signum(a);
  int d = signum(b);
  printf("signum %d = %d\n", a, c);  
  printf("signum %.2f = %d\n", b, d);

  char e = 'A';
  int f = signum(e);
  printf("signum of letter %c = %d\n", e, f);
  
  int *dev_a;
  hipMallocManaged(&dev_a, sizeof(int));
  dev_a[0] = a;
  signumCaller<<<BLOCKS, THREADS>>>(dev_a);
  hipDeviceSynchronize();
  printf("kernel signum %d = %d\n", a, dev_a[0]);
  hipFree(dev_a);
  return 0;
}