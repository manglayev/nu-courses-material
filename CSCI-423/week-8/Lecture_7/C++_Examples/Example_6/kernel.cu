#include "hip/hip_runtime.h"
#include "kernel_header.cuh"

__global__ void globalFunction(int *dev_a)
{
  int thread = threadIdx.x + blockIdx.x*blockDim.x;
  if(thread < THREADS)
  {
    #ifdef SQUARE
      dev_a[thread] = dev_a[thread]*dev_a[thread];
    #endif
    #ifdef CUBE
      dev_a[thread] = deviceFunction(dev_a[thread]);
    #endif
  }
}

void wrapper()
{
  #ifdef SQUARE
  int a[THREADS] = { 4, -5, 6};
  printf("SQUARE\n");
  #endif
  #ifdef CUBE
  int a[THREADS] = { -1, 2, 3 };
  printf("CUBE\n");
  #endif
  printf("START\n");
  for(int i = 0; i<THREADS; i++)
    printf("a[%d] = %.2d; ", i, a[i]);
  printf("\nEND\n");

  int *dev_a;
  hipMalloc((void**)&dev_a, THREADS*sizeof(int));
  hipMemcpy(dev_a, a, THREADS*sizeof(int), hipMemcpyHostToDevice);
  globalFunction<<<BLOCKS, THREADS>>>(dev_a);
  hipMemcpy(a, dev_a, THREADS*sizeof(int), hipMemcpyDeviceToHost);

  for(int i = 0; i<THREADS; i++)
    printf("a[%d] = %.2d; ", i, a[i]);
  printf("\n");
  hipFree(dev_a);
}
