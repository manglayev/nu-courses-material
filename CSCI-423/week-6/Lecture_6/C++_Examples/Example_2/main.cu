#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits>
using namespace std;

#define THREADS 5
#define BLOCKS 1

typedef float orange;

__global__ void testFunction(float *dev_a, float *dev_b, orange *dev_c)
{
  int thread = threadIdx.x;
  if(thread < THREADS)
  {
    dev_a[thread] = dev_a[thread] + dev_b[0] + dev_c[0];
  }
}

int main()
{
  printf("\nMAIN START\n");
  float a[THREADS] = { 1, 2, 3, 4, 5 };
  float *dev_a;
  hipMalloc((void**)&dev_a, THREADS*sizeof(float));
  hipMemcpy(dev_a, a, THREADS*sizeof(float), hipMemcpyHostToDevice);

  float b = 25;
  float *dev_b;
  hipMalloc((void**)&dev_b, sizeof(float));
  hipMemcpy(dev_b, &b, sizeof(float), hipMemcpyHostToDevice);

  orange c = 75;
  orange *dev_c;
  hipMalloc((void**)&dev_c, sizeof(orange));
  hipMemcpy(dev_c, &c, sizeof(orange), hipMemcpyHostToDevice);

  testFunction<<<BLOCKS, THREADS>>>(dev_a, dev_b, dev_c);

  hipMemcpy(a, dev_a, THREADS*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&b, dev_b, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&c, dev_c, sizeof(orange), hipMemcpyDeviceToHost);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  printf("\nAFTER CUDA FREE\n");
  for(int i = 0; i<THREADS; i++)
  {
    printf("a[%d] = %.2f; ", i, a[i]);
  }
  printf("\n");
  printf("b = %.2f; \n", b);
  printf("c = %.2f; \n", c);
  printf("\nEND\n");

  return 0;
}
