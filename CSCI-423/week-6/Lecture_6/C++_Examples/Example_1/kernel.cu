#include "hip/hip_runtime.h"
#include "kernel_header.cuh"

#include <hip/hip_runtime.h>
#include ""
#include "hip/hip_runtime.h"

__global__ void cudaFunction(int *d)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if(index < 1)
  {
    printf("1. d[%d] = %d\n", 0, d[0]);
    d[0] = d[0] * 3;
    printf("2. d[%d] = %d\n", 0, d[0]);
  }
}

void wrapper(int c)
{
  printf("BEFORE:\n");
  printf("c = %d;\n", c);

  int *dev_c;
  hipMalloc((void**)&dev_c, sizeof(int));
  hipMemcpy(dev_c, &c, sizeof(int), hipMemcpyHostToDevice);

  cudaFunction<<<BLOCKS, THREADS>>>(dev_c);
  hipError_t err_1;
  err_1 = hipGetLastError();
  if (err_1 != hipSuccess)
  {
      printf("Error: %s\n", hipGetErrorString(err_1));
  }
  hipDeviceSynchronize();
  hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(dev_c);
  printf("AFTER:\n");
  printf("c = %d;\n", c);
}
