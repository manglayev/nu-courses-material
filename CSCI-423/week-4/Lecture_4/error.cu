#include <hip/hip_runtime.h>
#include <iostream>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, const char* const func, const char* const file, const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char* const file, const int line)
{
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

int main()
{
    float* p;
    // This will produce error.
    CHECK_CUDA_ERROR(hipMalloc(&p, 1000000000000000 * sizeof(float)));
    // This will be successful.
    CHECK_CUDA_ERROR(hipMalloc(&p, 10 * sizeof(float)));
    // This will be successful.
    CHECK_CUDA_ERROR(hipFree(p));
    // The last error still has not been reset here.
    // This will produce the same error as
    // cudaMalloc(&p, 1000000000000000 * sizeof(float))
    CHECK_LAST_CUDA_ERROR();
    // The last error has been reset here.
    CHECK_LAST_CUDA_ERROR();
}