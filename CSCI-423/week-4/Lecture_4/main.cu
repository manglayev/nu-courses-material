
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
  /*
   * Device ID is required first to query the device.
   */

  int deviceId;
  hipGetDevice(&deviceId);

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);

  /*
   * `props` now contains several properties about the current device.
   */

  int computeCapabilityMajor = props.major;
  int computeCapabilityMinor = props.minor;
  int multiProcessorCount = props.multiProcessorCount;
  int warpSize = props.warpSize;

    /*
    int mgs[] = {props.maxGridSize[3]};
    for(int a = 0; a < 3; a++)
    {
        printf("value %d = %d;\n",a, mgs[a]);
    }
    */

    int mtd[] = {props.maxThreadsDim[3]};
    for(int a = 0; a < 3; a++)
    {
        printf("value %d = %d;\n",a, mtd[a]);
    }

    int  mtpb = props.maxThreadsPerBlock;
    printf("max threads per block = %d;\n",mtpb);
    //16
    int mbpmp = props.maxBlocksPerMultiProcessor;
    printf("max blocks per sm = %d;\n",mbpmp);
    //16

    int mtpmp = props.maxThreadsPerMultiProcessor;
    printf("max threads per sm = %d;\n",mtpmp);

  printf("\nDevice ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n", deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize);
}