#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>
using namespace std;

template<typename T> T signum(T x)
{
  if(x > 0)
    return 1;
  else if(x < 0)
    return -1;
  else
    return 0;
}

int main()
{
  int a = 5;
  float b = -19.234;
  int c = signum(a);
  int d = signum(b);
  printf("signum %d = %d\n", a, c);  
  printf("signum %.2f = %d\n", b, d);  
  return 0;
}