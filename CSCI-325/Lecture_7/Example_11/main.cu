#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>

#define THREADS 5
#define BLOCKS 1

#define CUDA_HOSTDEV __host__ __device__

template <typename T> class Array
{
  private:
	  T* ptr;
	  int size;

  public:
    CUDA_HOSTDEV Array(T arr[], int s);
    CUDA_HOSTDEV void print();
};

template <class T> CUDA_HOSTDEV Array<T>::Array(T arr[], int s)
{
	ptr = new T[s];
	size = s;
	for (int i = 0; i < size; i++)
		ptr[i] = arr[i];
}

template <class T> CUDA_HOSTDEV void Array<T>::print()
{
    for (int i = 0; i < size; i++)
      printf(" %d", *(ptr + i));
    printf("\n");
}

__global__ void globalFunction(int *x)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx < 1)
  {
    int arr[THREADS] = { 1, 2, 3, 4, 5 };
    Array<int> array(arr, THREADS);
    printf("PRINT FROM KERNEL:\n");
    array.print();
  }
}

int main()
{
  //HOST
  int arr[THREADS] = { 1, 2, 3, 4, 5 };
  Array<int> array(arr, THREADS);
  printf("PRINT FROM HOST:\n");
  array.print();
  //KERNEL
  int *dev_a;
  hipMallocManaged(&dev_a, sizeof(int));
  dev_a[0] = THREADS;
  globalFunction<<<BLOCKS, THREADS>>>(dev_a);
  hipDeviceSynchronize();
  printf("AFTER KERNEL %d = %d\n", THREADS, dev_a[0]);
  hipFree(dev_a);
  return 0;
}