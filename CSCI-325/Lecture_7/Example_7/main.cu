#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>

#define THREADS 1
#define BLOCKS 1

enum enumOrder {h1, h2, h3, h4};

__global__ void globalFunction(int a, int *b, enumOrder order)
{
  int thread = threadIdx.x + blockIdx.x*blockDim.x;
  if(thread < THREADS)
  {
    switch(order)
    {
      case h1:
        b[0] = a+a;
        break;
      case h2:
        b[0] = a-a;
        break;
      case h3:
        b[0] = a*a;
        break;
      default:
        b[0] = a/a;
        break;
    }
  }
}

int main()
{
  int a = 4;
  int *b;
  hipMallocManaged(&b, sizeof(int));
  b[0] = 2;
  globalFunction<<<BLOCKS, THREADS>>>(a, b, h1);
  hipDeviceSynchronize();
  printf("b = %d;\n", b[0]);
  hipFree(b);
  return 0;
}
