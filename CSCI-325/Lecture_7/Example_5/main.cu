#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>

#define THREADS 1
#define BLOCKS 1

__device__ int square(float numberInSquare)
{
  return numberInSquare * numberInSquare;
}

__device__ void cube(float numberInCube)
{
  numberInCube = square(numberInCube) * numberInCube;
  printf("numberInCube     = %.2f;\n", numberInCube);
}

__global__ void testFunction(float numberInGlobal)
{
  int thread = threadIdx.x;
  if(thread == 0)
  {
    printf("numberInGlobal 1 = %.2f;\n", numberInGlobal);
    numberInGlobal = square(numberInGlobal);
    printf("numberInGlobal 2 = %.2f;\n", numberInGlobal);
    cube(numberInGlobal);
    printf("numberInGlobal 3 = %.2f;\n", numberInGlobal);
  }
}

int main()
{
  float dev_a = 2;
  testFunction<<<BLOCKS, THREADS>>>(dev_a);
  hipDeviceSynchronize();
  printf("dev_a after kernel: = %.2f;\n", dev_a);
  return 0;
}
