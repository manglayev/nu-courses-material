#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>

#define THREADS 1
#define BLOCKS 1

__host__ __device__ int customFunction(int a, int *b)
{
  return a+a;
}

__global__ void globalFunction(int a, int *b)
{
  int thread = threadIdx.x + blockIdx.x*blockDim.x;
  if(thread < THREADS)
  {
    b[0] = customFunction(a, b)+a;
  }
}

int main()
{
  int a = 4;
  int *b;
  hipMallocManaged(&b, sizeof(int));
  b[0] = customFunction(a, b);
  printf("b 1 = %d;\n", b[0]);
  globalFunction<<<BLOCKS, THREADS>>>(a, b);
  hipDeviceSynchronize();
  printf("b 2 = %d;\n", b[0]);
  hipFree(b);
  return 0;
}