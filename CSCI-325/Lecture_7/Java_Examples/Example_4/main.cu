#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>

#define THREADS 5
#define BLOCKS 1

__global__ void testFunction(float *dev_a, float *dev_b, float *dev_c, float dev_d)
{
  int thread = threadIdx.x;
  if(thread == 0)
  {
    printf("dev[%d] = %.2f;\n", thread, dev_a[thread]);
    printf("b = %.2f;\n", *dev_b);
    printf("c 1 = %.2f;\n", *dev_c);
    *dev_c = 51.1*2;
    printf("c 2 = %.2f;\n", *dev_c);
    dev_d = dev_d*3;
    printf("d 2 = %.2f;\n", dev_d);
  }
}

int main()
{
  float a[THREADS] = { 1, 2, 3, 4, 5 };
  printf("BEFORE START\n");
  for(int i = 0; i<THREADS; i++)
  {
    printf("a[%d] = %.2f; ", i, a[i]);
  }
  printf("\nBEFORE END\n");
  float *dev_a;
  hipMalloc((void**)&dev_a, THREADS*sizeof(float));
  hipMemcpy(dev_a, a, THREADS*sizeof(float), hipMemcpyHostToDevice);

  float b = 25;
  float *dev_b;
  hipMalloc((void**)&dev_b, sizeof(float));
  hipMemcpy(dev_b, &b, sizeof(float), hipMemcpyHostToDevice);

  float *dev_c;
  hipMallocManaged(&dev_c, sizeof(float));

  float d = 77;
  testFunction<<<BLOCKS, THREADS>>>(dev_a, dev_b, dev_c, d);
  hipDeviceSynchronize();
  printf("dev_c after kernel: = %.2f;\n", *dev_c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  printf("d after kernel free: = %.2f;\n", d);
  return 0;
}
