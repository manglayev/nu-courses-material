#include "hip/hip_runtime.h"
#include "kernel_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define customDefinedFunction(a, b)(a*b/(THREADS));
void fillArray(float *b);
using namespace std;

__device__ int deviceFunction(int a, int b)
{
  int c = a - b;
  int d = b - a;
  return max(c,d);
}

__global__ void globalFunction(float *dev_b)
{
  int index = threadIdx.x + blockIdx.x*blockDim.x;
  if(index == 0)
  {
    float a = customDefinedFunction(4, 16);
    printf("%.2f\n", a);
    int b = deviceFunction(4, 16);
    printf("%d\n", b);
    dev_b[index] = a + b;
  }
}
void wrapper()
{
  float *b = new float[CUDASIZE];
  fillArray(b);
  float *dev_b;
  hipMalloc((void**)&dev_b, CUDASIZE*sizeof(float));
	hipMemcpy(dev_b, b, CUDASIZE*sizeof(float), hipMemcpyHostToDevice);
  globalFunction<<<BLOCKS, THREADS>>>(dev_b);
  hipMemcpy(b, dev_b, CUDASIZE*sizeof(float), hipMemcpyDeviceToHost);
  for(int a=0; a<CUDASIZE; a++)
  {
    printf("b[%d] = %.2f\n", a, b[a]);
  }
  hipFree(dev_b);
}//end wrapper function

void fillArray(float *b)
{
  for(int a=0; a<CUDASIZE; a++)
  {
    b[a] = a+1;
  }
}
